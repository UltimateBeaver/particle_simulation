#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "common.h"

#define NUM_THREADS 256

extern double size;

// calculate particle's bin number
int binNum(particle_t &p, int bpr)
{
    return ( floor(p.x/cutoff) + bpr*floor(p.y/cutoff) );
}

//
//  benchmarking program
//
int main( int argc, char **argv )
{
    int navg,nabsavg=0;
    double dmin, davg, absmin= 1.0,absavg =0.0;

    if( find_option( argc, argv, "-h" ) >=0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        printf( "-s <filename> to specify a summary file name\n" );
        printf( "-no turns off all correctness checks and particle output\n");
        return 0;
    }

    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    char *sumname = read_string( argc, argv, "-s", NULL );

    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    FILE *fsum = sumname ? fopen ( sumname, "a" ) : NULL;

    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );
    set_size( n );
    init_particles( n, particles );

    // create spatial bins (of size cutoff by cutoff)
    double size = sqrt( density*n );
    int bpr = ceil(size/cutoff);
    int numbins = bpr*bpr;
    vector<particle_t*> *bins = new vector<particle_t*>[numbins];

    //
    //  simulate a number of time steps
    //
    double simulation_time = read_timer( );
    for( int step = 0; step < NSTEPS; step++ )
    {
      navg = 0;
      davg = 0.0;
      dmin = 1.0;

      // clear bins at each time step
      for (int m = 0; m < numbins; m++)
        bins[m].clear();

      // place particles in bins
      for (int i = 0; i < n; i++)
        bins[binNum(particles[i],bpr)].push_back(particles + i);

      //
      //  compute forces
      //
      for( int p = 0; p < n; p++ )
      {
        particles[p].ax = particles[p].ay = 0;

        // find current particle's bin, handle boundaries
        int cbin = binNum( particles[p], bpr );
        int lowi = -1, highi = 1, lowj = -1, highj = 1;
        if (cbin < bpr)
          lowj = 0;
        if (cbin % bpr == 0)
          lowi = 0;
        if (cbin % bpr == (bpr-1))
          highi = 0;
        if (cbin >= bpr*(bpr-1))
          highj = 0;

        // apply nearby forces
        for (int i = lowi; i <= highi; i++)
          for (int j = lowj; j <= highj; j++)
          {
            int nbin = cbin + i + bpr*j;
            for (int k = 0; k < bins[nbin].size(); k++ )
              apply_force( particles[p], *bins[nbin][k], &dmin, &davg, &navg);
          }
      }

      //
      //  move particles
      //
      for( int p = 0; p < n; p++ )
        move( particles[p] );

    }
    simulation_time = read_timer( ) - simulation_time;

    printf( "n = %d, simulation time = %g seconds", n, simulation_time);

    //
    // Printing summary data
    //
    if( fsum)
        fprintf(fsum,"%d %g\n",n,simulation_time);

    //
    // Clearing space
    //
    if( fsum )
        fclose( fsum );
    free( particles );
    delete [] bins;
    if( fsave )
        fclose( fsave );

    return 0;
}
